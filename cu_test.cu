#include <string>
#include <iostream>
#include "src/gpu_match.cuh"

using namespace std;
using namespace STMatch;

#define TIMEOUT_QUEUE_CAP 1'000'000

int main(int argc, char* argv[]) {

  hipSetDevice(0);


  STMatch::GraphPreprocessor g(argv[1]);

  std::cout << "Before PatternPreprocessor" << std::endl;
  
  STMatch::PatternPreprocessor p(argv[2]);
  
  std::cout << "Before build_src_vtx" << std::endl;

  g.build_src_vtx(p);

  std::cout << "after build_src_vtx" << std::endl;


  std::cout << "conditions: " << std::endl;
  for (int i = 0; i < p.order_.size(); i++) 
  {
      std::cout << i << ": ";
      for (int j = 0; j < p.order_[i].size(); j++)
          std::cout << GetCondOperatorString(p.order_[i][j].first) << "(" << p.order_[i][j].second << "), ";
      std::cout << std::endl;
  }

  // copy graph and pattern to GPU global memory
  Graph* gpu_graph = g.to_gpu();
  Pattern* gpu_pattern = p.to_gpu();
  JobQueue* gpu_queue = JobQueuePreprocessor(g.g, p).to_gpu();
  // JobQueue* gpu_queue = nullptr;
  CallStack* gpu_callstack;

  // allocate the callstack for all warps in global memory
  graph_node_t* slot_storage;
  hipMalloc(&slot_storage, sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * GRAPH_DEGREE);
  //cout << "global memory usage: " << sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE / 1024.0 / 1024 / 1024 << " GB" << endl;

  std::vector<CallStack> stk(NWARPS_TOTAL);

  for (int i = 0; i < NWARPS_TOTAL; i++) {
    auto& s = stk[i];
    memset(s.iter, 0, sizeof(s.iter));
    memset(s.slot_size, 0, sizeof(s.slot_size));
    s.slot_storage = (graph_node_t(*)[GRAPH_DEGREE])((char*)slot_storage + i * sizeof(graph_node_t) * MAX_SLOT_NUM * GRAPH_DEGREE);
  }
  hipMalloc(&gpu_callstack, NWARPS_TOTAL * sizeof(CallStack));
  hipMemcpy(gpu_callstack, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);

  size_t* gpu_res;
  hipMalloc(&gpu_res, sizeof(size_t) * NWARPS_TOTAL);
  hipMemset(gpu_res, 0, sizeof(size_t) * NWARPS_TOTAL);
  size_t* res = new size_t[NWARPS_TOTAL];

  int* idle_warps;
  hipMalloc(&idle_warps, sizeof(int) * GRID_DIM);
  hipMemset(idle_warps, 0, sizeof(int) * GRID_DIM);

  int* idle_warps_count;
  hipMalloc(&idle_warps_count, sizeof(int));
  hipMemset(idle_warps_count, 0, sizeof(int));

  int* global_mutex;
  hipMalloc(&global_mutex, sizeof(int) * GRID_DIM);
  hipMemset(global_mutex, 0, sizeof(int) * GRID_DIM);

  bool* stk_valid;
  hipMalloc(&stk_valid, sizeof(bool) * GRID_DIM);
  hipMemset(stk_valid, 0, sizeof(bool) * GRID_DIM);

  int* gpu_timeout_queue_space;
  hipMalloc(&gpu_timeout_queue_space, sizeof(int) * TIMEOUT_QUEUE_CAP * (STOP_LEVEL + 1));
  Queue* gpu_timeout_queue;
  hipMallocManaged(&gpu_timeout_queue, sizeof(Queue));
  gpu_timeout_queue->queue_ = gpu_timeout_queue_space;
  gpu_timeout_queue->size_ = TIMEOUT_QUEUE_CAP * (STOP_LEVEL + 1);
  gpu_timeout_queue->resetQueue();
  
  // timer starts here
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  //cout << "shared memory usage: " << sizeof(Graph) << " " << sizeof(Pattern) << " " << sizeof(JobQueue) << " " << sizeof(CallStack) * NWARPS_PER_BLOCK << " " << NWARPS_PER_BLOCK * 33 * sizeof(int) << " Bytes" << endl;

  _parallel_match << <GRID_DIM, BLOCK_DIM >> > (gpu_graph, gpu_pattern, gpu_callstack, gpu_queue, gpu_res, idle_warps, 
                                              idle_warps_count, global_mutex, gpu_timeout_queue);


  hipEventRecord(stop);
  
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  //printf("matching time: %f ms\n", milliseconds);

  hipMemcpy(res, gpu_res, sizeof(size_t) * NWARPS_TOTAL, hipMemcpyDeviceToHost);

  unsigned long long tot_count = 0;
  for (int i=0; i<NWARPS_TOTAL; i++) tot_count += res[i];

  if(!LABELED) tot_count = tot_count * p.PatternMultiplicity;
  
  printf("%s\t%f\t%llu\n", argv[2], milliseconds, tot_count);
  //cout << "count: " << tot_count << endl;
  return 0;
}
